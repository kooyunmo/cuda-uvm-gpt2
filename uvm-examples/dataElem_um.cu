
#include <hip/hip_runtime.h>
#include <string.h>
#include <stdio.h>

struct DataElement
{
  char *name;
  int value;
};

__global__
void Kernel(DataElement *elem) {
  printf("On device: name=%s, value=%d\n", elem->name, elem->value);
  
  elem->name[0] = 'd';
  elem->value++;
}

void launch(DataElement *elem) {
  Kernel<<< 1, 1 >>>(elem);
  hipDeviceSynchronize();
}

int main(void)
{
  DataElement *e;
  hipMallocManaged((void **)&e, sizeof(DataElement));

  e->value = 10;
  hipMallocManaged((void **)&(e->name), sizeof(char) * (strlen("hello") + 1));
  strcpy(e->name, "hello");

  launch(e);

  printf("On host: name=%s, value=%d\n", e->name, e->value);
  
  hipFree(e->name);
  hipFree(e);

  hipDeviceReset();
}

